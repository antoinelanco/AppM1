#include "hip/hip_runtime.h"
#include "test.h"
#include "cudautils.h"


__global__ void init_vec(float* vec) {
	int tid = threadIdx.x;
	vec[tid] = 1.0f;
}

/*int getNbThreadPerBlock(int device) {
	int value;
	hipDeviceGetAttribute(&value, hipDeviceAttributeMaxThreadsPerBlock, device);
	return value;
}*/

float* mallocCuda(int nbElt) {
	float* res;
	hipMalloc(&res, sizeof(float) * nbElt);
	return res;
}

float* copyToHost(float* deviceData, int nbElt) {
	float* res = (float*) malloc(sizeof(float) * nbElt);
	hipMemcpy(res, deviceData, sizeof(float) * nbElt, hipMemcpyDeviceToHost);
	return res;
}

void initVec(float* deviceData, int nbElt) {
	int nbThread = getNbThreadPerBlock(0);
	init_vec<<<1, nbElt>>>(deviceData);
}