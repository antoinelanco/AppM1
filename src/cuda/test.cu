#include "hip/hip_runtime.h"
#include "test.h"


__global__ void init_vec(float* vec) {
	int tid = threadIdx.x;
	vec[tid] = 15.;
}

int getNbThreadPerBlock(int device) {
	int value;
	hipDeviceGetAttribute(&value, hipDeviceAttributeMaxThreadsPerBlock, device);
	return value;
}

float* mallocCuda(int nbElt) {
	float* res;
	hipMalloc(&res, sizeof(float) * nbElt);
	return res;
}

float* copyToHost(float* deviceData, int nbElt) {
	float* res = (float*) malloc(sizeof(float) * nbElt);
	hipMemcpy(res, deviceData, sizeof(float) * nbElt, hipMemcpyDeviceToHost);
	return res;
}

void initVec(float* deviceData, int nbElt) {
	int nbThread = getNbThreadPerBlock(0);
	init_vec<<<1, nbElt>>>(deviceData);
}