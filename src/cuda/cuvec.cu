#include "hip/hip_runtime.h"
#include "cuvec.h"
#include "cudautils.h"
#include <stdlib.h>
#include <iostream>

using namespace std;

__global__ void reduce0(float* g_odata, float* g_idata1, float* g_idata2) {
	extern __shared__ float sdata[];
	// each thread loads one element from global to shared mem

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	sdata[tid] = g_idata1[i] * g_idata2[i];
	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	// write result for this block to global mem
	if (tid == 0) {
		g_odata[blockIdx.x] = sdata[0];
		//atomicAdd(g_odata, sdata[0]);
	}
}

// https://stackoverflow.com/questions/26853363/dot-product-for-dummies-with-cuda-c
__global__ void dotCuda3(float *a, float *b, float *c){
	__shared__ float cache[1024];
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int cacheIndex = threadIdx.x;
	float temp = a[tid] * b[tid];//+ a[tid + blockDim.x] * b[tid + blockDim.x];
	cache[cacheIndex] = temp;
	__syncthreads();

	for (unsigned int i = blockDim.x >> 1; i > 0; i >>= 1) {
		if (cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];
		__syncthreads();
	}

	if (cacheIndex == 0){
		c[blockIdx.x] = cache[0];
	}
}

__global__ void dotCuda(float* tmp, float* t1, float* t2, int size) {
	//unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	tmp[i] = t1[i] * t2[i];
	__syncthreads();

	int mididx = size / 2;

	while (i < mididx) {
		tmp[i] += tmp[i + mididx];
		mididx /= 2;
		__syncthreads();
	}
	//atomicAdd(tmp, p);
}

__global__ void init_vec(float* vec, float value) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	vec[tid] = value;
}

CudaVec::CudaVec(int size) {
	this->size = size;
	if (this->size % 2 != 0) {
		cout << "Must me multiple of 2 !" << endl;
		exit(0);
	}
	hipMalloc(&this->cudaptr, sizeof(float) * this->size);
	hipMemset(this->cudaptr, 0, sizeof(float) * this->size);
}

CudaVec::CudaVec(float* hostData, int size) {
	this->size = size;
	if (this->size % 2 != 0) {
		cout << "Must me multiple of 2 !" << endl;
		exit(0);
	}
	hipMalloc(&this->cudaptr, sizeof(float) * this->size);
	hipMemcpy(this->cudaptr, hostData, sizeof(float) * this->size, hipMemcpyHostToDevice);
}

void CudaVec::fill(float value) {
	int nbBlX = getNbBlockDimX(0);
	int thrPBl = getNbThreadPerBlock(0);

	if (this->size <= thrPBl) {
		init_vec<<<1, this->size>>>(this->cudaptr, value);
		return;
	}

	int neededBl = this->size / nbBlX;
	if (neededBl <= nbBlX) {
		init_vec<<<neededBl, thrPBl>>>(this->cudaptr, value);
		return;
	}
	cout << "Unimplemented nbBlNeeded : " << neededBl << " (" << nbBlX << " available)" << endl;
	exit(0);
}

void CudaVec::free() {
	hipFree(this->cudaptr);
}

float* CudaVec::toHost() {
	float* hostVec = (float*) malloc(sizeof(float) * this->size);
	hipMemcpy(hostVec, this->cudaptr, sizeof(float) * this->size, hipMemcpyDeviceToHost);
	return hostVec;
}

int CudaVec::getSize() {
	return this->size;
}

float* CudaVec::dot(CudaVec other, int subSize) {
	if (this->size != other.size) {
		cout << "Uncompatible size !" << endl;
		exit(0);
	}

	int nbBlX = getNbBlockDimX(0);
	int thrPBl = getNbThreadPerBlock(0);

	if (subSize % thrPBl != 0) {
		cout << "Error in subSize !" << endl;
		exit(0);
	}

	int neededBl = this->size / thrPBl;

	float* tmp;
	hipMalloc(&tmp, sizeof(float) * neededBl);

	if (this->size > thrPBl && neededBl <= nbBlX) {
		//Dev_dot<<<neededBl, thrPBl>>>(this->cudaptr, other.cudaptr, tmp, this->size);
		//dotCuda<<<neededBl, thrPBl>>>(tmp, this->cudaptr, other.cudaptr, this->size);
		//reduce0<<<neededBl, thrPBl>>>(tmp, this->cudaptr, other.cudaptr);
		dotCuda3<<<neededBl, thrPBl>>>(this->cudaptr, other.cudaptr, tmp);
	} else {
		cout << "Bug" << endl;
		exit(0);
	}

	float* result = new float[neededBl];
	hipMemcpy(result, tmp, sizeof(float) * neededBl, hipMemcpyDeviceToHost);
	hipFree(tmp);

	int nbRes = this->size / subSize;
	float* sum = new float[nbRes];
	int nb = neededBl / nbRes;
	cout << "nbRes " << nbRes << ", nb " << nb << endl;
	for (int i = 0; i < nbRes; i++) {
		for(int j = 0; j < nb; j++) {
			sum[i] += result[i * nb + j];
		}
	}
	delete[] result;
	return sum;
	/*float sum = 0.f;
	for (int i = 0; i < neededBl; i++) {
	sum += result[i];
}

return sum;*/
/*float result = 0.f;
hipMemcpy(&result, tmp, sizeof(float) * 1, hipMemcpyDeviceToHost);
hipFree(tmp);
return result;*/
}
