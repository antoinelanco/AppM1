#include "hip/hip_runtime.h"
#include "cuvec.h"
#include "cudautils.h"
#include <stdlib.h>
#include <iostream>

using namespace std;

__global__ void init_vec(float* vec, float value) {
	int tid = blockIdx.x *blockDim.x + threadIdx.x;
	vec[tid] = value;
}

CudaVec::CudaVec(int size) {
	this->size = size;
	if (this->size % 32 != 0)
		exit(0);
	hipMalloc(&this->cudaptr, sizeof(float) * this->size);
	hipMemset(this->cudaptr, 0, sizeof(float) * this->size);
}

void CudaVec::fill(float value) {
	int nbBlX = getNbBlockDimX(0);
	int thrPBl = getNbThreadPerBlock(0);

	if (this->size <= thrPBl) {
		init_vec<<<1, this->size>>>(this->cudaptr, value);
		return;
	}

	int neededBl = this->size / nbBlX;
	if (neededBl <= nbBlX) {
		init_vec<<<neededBl, thrPBl>>>(this->cudaptr, value);
		return;
	}
	cout << "Unimplemented nbBlNeeded : " << neededBl << " (" << nbBlX << " available)" << endl;
	exit(0);
}

void CudaVec::free() {
	hipFree(this->cudaptr);
}

float* CudaVec::toHost() {
	float* hostVec = (float*) malloc(sizeof(float) * this->size);
	hipMemcpy(hostVec, this->cudaptr, sizeof(float) * this->size, hipMemcpyDeviceToHost);
	return hostVec;
}

int CudaVec::getSize() {
	return this->size;
}