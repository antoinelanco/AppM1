#include "hip/hip_runtime.h"
#include "cuvec.h"
#include "cudautils.h"
#include <stdlib.h>
#include <iostream>

using namespace std;

__global__ void reduce0(float* g_odata, float* g_idata1, float* g_idata2) {
	extern __shared__ float sdata[];
	// each thread loads one element from global to shared mem

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	sdata[tid] = g_idata1[i] * g_idata2[i];
	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	// write result for this block to global mem
	if (tid == 0) { 
		g_odata[blockIdx.x] = sdata[0];
		//atomicAdd(g_odata, sdata[0]);
	}
}

// https://stackoverflow.com/questions/26853363/dot-product-for-dummies-with-cuda-c
__global__ void dotCuda3(float *a, float *b, float *c){
	__shared__ float cache[1024];
	//int tid = threadIdx.x + blockIdx.x*blockDim.x; 
	int tid = blockIdx.x * (blockDim.x*2) + threadIdx.x;
	int cacheIndex = threadIdx.x; 
	float temp = a[tid] * b[tid] + a[tid + blockDim.x] * b[tid + blockDim.x];
	cache[cacheIndex] = temp; 
	__syncthreads(); 

	for (unsigned int i = blockDim.x >> 1; i > 0; i >>= 1) {
    	if(cacheIndex < i)
        	cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();      
	}

	if (cacheIndex == 0){ 
    	c[blockIdx.x] = cache[0]; 
	}
}

__global__ void dotCuda(float* tmp, float* t1, float* t2, int size) {
	//unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	tmp[i] = t1[i] * t2[i];
	__syncthreads();

	int mididx = size / 2;

	while (i < mididx) {
		tmp[i] += tmp[i + mididx];
		mididx /= 2;
		__syncthreads();
	}
	//atomicAdd(tmp, p);
}

__global__ void init_vec(float* vec, float value) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	vec[tid] = value;
}

CudaVec::CudaVec(int size) {
	this->size = size;
	if (this->size % 2 != 0)
		exit(0);
	hipMalloc(&this->cudaptr, sizeof(float) * this->size);
	hipMemset(this->cudaptr, 0, sizeof(float) * this->size);
}

void CudaVec::fill(float value) {
	int nbBlX = getNbBlockDimX(0);
	int thrPBl = getNbThreadPerBlock(0);

	if (this->size <= thrPBl) {
		init_vec<<<1, this->size>>>(this->cudaptr, value);
		return;
	}

	int neededBl = this->size / nbBlX;
	if (neededBl <= nbBlX) {
		init_vec<<<neededBl, thrPBl>>>(this->cudaptr, value);
		return;
	}
	cout << "Unimplemented nbBlNeeded : " << neededBl << " (" << nbBlX << " available)" << endl;
	exit(0);
}

void CudaVec::free() {
	hipFree(this->cudaptr);
}

float* CudaVec::toHost() {
	float* hostVec = (float*) malloc(sizeof(float) * this->size);
	hipMemcpy(hostVec, this->cudaptr, sizeof(float) * this->size, hipMemcpyDeviceToHost);
	return hostVec;
}

int CudaVec::getSize() {
	return this->size;
}

float CudaVec::dot(CudaVec other) {
	if (this->size != other.size) {
		cout << "Uncompatible size !" << endl;
		exit(0);
	}
	//tmp[0] = 0;

	//int fake_size = (int) pow(2, ceil(log(this->size)/log(2)));

	int nbBlX = getNbBlockDimX(0);
	int thrPBl = getNbThreadPerBlock(0);

	int neededBl = this->size / thrPBl / 2;

	float* tmp;
	hipMalloc(&tmp, sizeof(float) * neededBl);
	hipMemset(tmp, 0, sizeof(float) * neededBl);

	if (this->size <= thrPBl) {
		//dotCuda<<<1, fake_size>>>(tmp, this->cudaptr, other.cudaptr, this->size);
		//Dev_dot<<<1, this->size>>>(this->cudaptr, other.cudaptr, tmp, this->size);
		//reduce0<<<1, this->size>>>(tmp, this->cudaptr, other.cudaptr);
		cout << "pas sur que ça marche ! 1 block of " << this->size << " threads" << endl;
		dotCuda3<<<1, this->size>>>(this->cudaptr, other.cudaptr, tmp);
	} else {
		
		cout << "pb appel, started : " << (neededBl * thrPBl) << ", curr : " << this->size << endl;

		if (neededBl <= nbBlX) {
			//Dev_dot<<<neededBl, thrPBl>>>(this->cudaptr, other.cudaptr, tmp, this->size);
			//dotCuda<<<neededBl, thrPBl>>>(tmp, this->cudaptr, other.cudaptr, this->size);
			//reduce0<<<neededBl, thrPBl>>>(tmp, this->cudaptr, other.cudaptr);
			dotCuda3<<<neededBl, thrPBl>>>(this->cudaptr, other.cudaptr, tmp);
		} else {
			cout << "Unimplemented nbBlNeeded : " << neededBl << " (" << nbBlX << " available)" << endl;
			exit(0);
		}
	}
	hipDeviceSynchronize();

	float* result = new float[neededBl];
	// Le resultat est dans tmp[0]
	hipMemcpy(result, tmp, sizeof(float) * neededBl, hipMemcpyDeviceToHost);
	hipFree(tmp);

	float sum = 0.f;
	for (int i = 0; i < neededBl; i++) {
		sum += result[i];
	}

	return sum;
	/*float result = 0.f;
	hipMemcpy(&result, tmp, sizeof(float) * 1, hipMemcpyDeviceToHost);
	hipFree(tmp);
	return result;*/
}