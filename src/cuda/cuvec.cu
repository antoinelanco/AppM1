#include "hip/hip_runtime.h"
#include "cuvec.h"
#include "cudautils.h"
#include <stdlib.h>
#include <iostream>

using namespace std;

__global__ void dotCuda(float* tmp, float* t1, float* t2, int size) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	tmp[tid] = t1[tid] * t2[tid];

	__syncthreads();

	int mididx = size / 2;

	while (tid <= mididx && mididx != 0) {
		tmp[tid] += tmp[tid * 2];
		mididx /= 2;
		__syncthreads();
	}
}

__global__ void init_vec(float* vec, float value) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	vec[tid] = value;
}

CudaVec::CudaVec(int size) {
	this->size = size;
	if (this->size % 32 != 0)
		exit(0);
	hipMalloc(&this->cudaptr, sizeof(float) * this->size);
	hipMemset(this->cudaptr, 0, sizeof(float) * this->size);
}

void CudaVec::fill(float value) {
	int nbBlX = getNbBlockDimX(0);
	int thrPBl = getNbThreadPerBlock(0);

	if (this->size <= thrPBl) {
		init_vec<<<1, this->size>>>(this->cudaptr, value);
		return;
	}

	int neededBl = this->size / nbBlX;
	if (neededBl <= nbBlX) {
		init_vec<<<neededBl, thrPBl>>>(this->cudaptr, value);
		return;
	}
	cout << "Unimplemented nbBlNeeded : " << neededBl << " (" << nbBlX << " available)" << endl;
	exit(0);
}

void CudaVec::free() {
	hipFree(this->cudaptr);
}

float* CudaVec::toHost() {
	float* hostVec = (float*) malloc(sizeof(float) * this->size);
	hipMemcpy(hostVec, this->cudaptr, sizeof(float) * this->size, hipMemcpyDeviceToHost);
	return hostVec;
}

int CudaVec::getSize() {
	return this->size;
}

float CudaVec::dot(CudaVec other) {
	if (this->size != other.size) {
		cout << "Uncompatible size !" << endl;
		exit(0);
	}

	float* tmp;
	hipMalloc(&tmp, sizeof(float) * this->size);

	

	int nbBlX = getNbBlockDimX(0);
	int thrPBl = getNbThreadPerBlock(0);

	if (this->size <= thrPBl) {
		dotCuda<<<1, this->size>>>(tmp, this->cudaptr, other.cudaptr, this->size);
	} else {
		int neededBl = this->size / nbBlX;
		if (neededBl <= nbBlX) {
			dotCuda<<<neededBl, thrPBl>>>(tmp, this->cudaptr, other.cudaptr, this->size);
		} else {
			cout << "Unimplemented nbBlNeeded : " << neededBl << " (" << nbBlX << " available)" << endl;
			exit(0);
		}
	}
	float result;
	hipMemcpy(&result, tmp, sizeof(float) * 1, hipMemcpyDeviceToHost);
	hipFree(tmp);
	return result;
}