#include "hip/hip_runtime.h"
#include "cuvec.h"
#include "cudautils.h"
#include <stdlib.h>
#include <iostream>

using namespace std;

__global__ void reduce0(float* g_odata, float* g_idata1, float* g_idata2) {
	extern __shared__ float sdata[];
	// each thread loads one element from global to shared mem

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	sdata[tid] = g_idata1[i] * g_idata2[i];
	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	// write result for this block to global mem
	if (tid == 0) { 
		g_odata[blockIdx.x] = sdata[0];
		//atomicAdd(g_odata, sdata[0]);
	}
}

__global__ void dotCuda(float* tmp, float* t1, float* t2, int size, int fake_size) {
	//unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	tmp[i] = t1[i] * t2[i];
	__syncthreads();

	int mididx = size / 2;

	//bool needCorrect = mididx % 2 == 1;

	while (i < mididx) {
		tmp[i] += tmp[i + mididx];
		mididx /= 2;
		__syncthreads();
	}
	//atomicAdd(tmp, p);
}

__global__ void init_vec(float* vec, float value) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	vec[tid] = value;
}

CudaVec::CudaVec(int size) {
	this->size = size;
	if (this->size % 2 != 0)
		exit(0);
	hipMalloc(&this->cudaptr, sizeof(float) * this->size);
	hipMemset(this->cudaptr, 0, sizeof(float) * this->size);
}

void CudaVec::fill(float value) {
	int nbBlX = getNbBlockDimX(0);
	int thrPBl = getNbThreadPerBlock(0);

	if (this->size <= thrPBl) {
		init_vec<<<1, this->size>>>(this->cudaptr, value);
		return;
	}

	int neededBl = this->size / nbBlX;
	if (neededBl <= nbBlX) {
		init_vec<<<neededBl, thrPBl>>>(this->cudaptr, value);
		return;
	}
	cout << "Unimplemented nbBlNeeded : " << neededBl << " (" << nbBlX << " available)" << endl;
	exit(0);
}

void CudaVec::free() {
	hipFree(this->cudaptr);
}

float* CudaVec::toHost() {
	float* hostVec = (float*) malloc(sizeof(float) * this->size);
	hipMemcpy(hostVec, this->cudaptr, sizeof(float) * this->size, hipMemcpyDeviceToHost);
	return hostVec;
}

int CudaVec::getSize() {
	return this->size;
}

float CudaVec::dot(CudaVec other) {
	if (this->size != other.size) {
		cout << "Uncompatible size !" << endl;
		exit(0);
	}
	//tmp[0] = 0;

	int fake_size = (int) pow(2, ceil(log(this->size)/log(2)));

	int nbBlX = getNbBlockDimX(0);
	int thrPBl = getNbThreadPerBlock(0);

	int neededBl = fake_size / nbBlX;

	float* tmp;
	hipMalloc(&tmp, sizeof(float) * neededBl);

	if (fake_size <= thrPBl) {
		dotCuda<<<1, fake_size>>>(tmp, this->cudaptr, other.cudaptr, this->size);
		//reduce0<<<1, this->size>>>(tmp, this->cudaptr, other.cudaptr);
	} else {
		
		//cout << "pb appel, started : " << (neededBl * thrPBl) << ", curr : " << this->size << endl;

		if (neededBl <= nbBlX) {
			dotCuda<<<neededBl, thrPBl>>>(tmp, this->cudaptr, other.cudaptr, this->size);
			//reduce0<<<neededBl, thrPBl>>>(tmp, this->cudaptr, other.cudaptr);
		} else {
			cout << "Unimplemented nbBlNeeded : " << neededBl << " (" << nbBlX << " available)" << endl;
			exit(0);
		}
	}

	/*float* result = (float*) malloc(sizeof(float) * neededBl);
	// Le resultat est dans tmp[0]
	hipMemcpy(result, tmp, sizeof(float) * neededBl, hipMemcpyDeviceToHost);
	hipFree(tmp);

	float sum = 0.f;
	for (int i = 0; i < neededBl; i++) {
		sum += result[i];
	}

	return sum;*/
	float result = 0.f;
	hipMemcpy(&result, tmp, sizeof(float) * 1, hipMemcpyDeviceToHost);
	hipFree(tmp);
	return result;
}