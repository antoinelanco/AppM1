
#include <hip/hip_runtime.h>
int getNbThreadPerBlock(int device) {
	int value;
	hipDeviceGetAttribute(&value, hipDeviceAttributeMaxThreadsPerBlock, device);
	return value;
}

int getNbBlockDimX(int device) {
	int value;
	hipDeviceGetAttribute(&value, hipDeviceAttributeMaxBlockDimX, device);
	return value;
}